#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <vector>
#include <thread>

#include <System/Util.h>
#include <Bodies/Particle.h>
#include <Common/Common.h>
#include <Maths/Vector2.h>


using Core::Maths::Vector::Vector2;
using namespace Core::Common;

// Enables the use of core classes to the device
#define __HIPCC__

//to be used to switch between the different methods
//#define NORMAL_PARALLELISATION

///-------------------------------------------------------------------------------------------------
/// <summary>	Calculates the force. </summary>
///
/// <remarks>	Adrian, 10/07/2017. </remarks>
///
/// <param name="p_currentBody">	[in,out] If non-null, the current body. </param>
/// <param name="p_otherBody">  	[in,out] If non-null, the other body. </param>
/// <param name="p_force">			[in,out] If non-null, the force. </param>
///-------------------------------------------------------------------------------------------------
__device__ void CalculateForce(Body* p_currentBody, Body* p_otherBody, Vector2* p_force)
{
	//direction of the force between the two bodies 
	Vector2 direction = p_currentBody->Position - p_otherBody->Position;
	// to avoid a singularity
	float distance = max(0.5f * (p_otherBody->Mass + p_currentBody->Mass), direction.Length());
	//calculate the 
	Vector2 tempForce = direction / (distance * distance * distance)  * (p_otherBody->Mass / 1);
	(*p_force) += tempForce;
}

///-------------------------------------------------------------------------------------------------
/// <summary>	Calculates the force. </summary>
///
/// <remarks>	Adrian, 10/07/2017. </remarks>
///
/// <param name="p_currentBody">	[in,out] If non-null, the current body. </param>
/// <param name="p_otherBody">  	[in,out] If non-null, the other body. </param>
///
/// <returns>	The calculated force. </returns>
///-------------------------------------------------------------------------------------------------
__device__ Vector2 CalculateForce(Body* p_currentBody, Body* p_otherBody)
{
	Vector2 direction = p_currentBody->Position - p_otherBody->Position;
	// avoids singularity
	float distance = max(0.5f * (p_otherBody->Mass + p_currentBody->Mass), direction.Length());
	Vector2 tempForce = direction / (distance * distance * distance)  * (p_otherBody->Mass / 1);
	return tempForce;
}

///-------------------------------------------------------------------------------------------------
/// <summary>	
/// 	Gets global index of a CUDA thread
/// </summary>
///
/// <remarks>	Adrian, 10/07/2017. </remarks>
///
/// <returns>	The global index. </returns>
///-------------------------------------------------------------------------------------------------
__device__ uint_t GetGlobalIndex()
{
	int blockDimension = blockDim.x * blockDim.y * blockDim.z;
	int blockInnerIndex = threadIdx.x + blockDim.x * threadIdx.y;
	int globalIndex = blockInnerIndex + gridDim.x * blockIdx.y * blockDimension + blockDimension * blockIdx.x;
	return globalIndex;
}


///-------------------------------------------------------------------------------------------------
/// <summary>
/// 	This method improves further on, by computing reducing the required computations (n^2)/2
/// </summary>
///
/// <remarks>	Adrian, 10/07/2017. </remarks>
///
/// <param name="p_bodies">			 	[in,out] If non-null, the bodies. </param>
/// <param name="p_decomposedForces">	[in,out] If non-null, the decomposed forces. </param>
/// <param name="p_deltaT">			 	The delta t. </param>
/// <param name="p_bodyCount">		 	Number of bodies. </param>
/// <param name="p_gTerm">			 	The term. </param>
///-------------------------------------------------------------------------------------------------
__global__ void ComputeIterationOptimised
		(Body* p_bodies, Vector2* p_decomposedForces , uint_t* p_startingPositions, int p_bodyCount ,float p_deltaT,  float p_gTerm)
{
	uint_t globalIndex = GetGlobalIndex();

	if (globalIndex < p_bodyCount - 1)
	{
		Body assignedBody = p_bodies[globalIndex];
		int matrixIndex = 0;
		int threadBodyCount = p_bodyCount;

		//find index in the matrix
		threadBodyCount -= 1;
		for (int count = 0; count < globalIndex; count++)
		{
			matrixIndex += threadBodyCount;
			threadBodyCount -= 1;
		}
		
		//save decomposedArray index starting position
		p_startingPositions[globalIndex] = matrixIndex;

		//fill the portion of forces at the specific indices and skip force computation with thy self
		for (int index = globalIndex+1; index < p_bodyCount; index++, matrixIndex++)
		{
			p_decomposedForces[matrixIndex] = CalculateForce(&assignedBody, &p_bodies[index]);
		}
	}
}

///-------------------------------------------------------------------------------------------------
/// <summary>	
/// 			Merge results (used for the secondary method)
/// </summary>
///
/// <remarks>	Adrian, 10/07/2017. </remarks>
///
/// <param name="p_bodies">			 	[in,out] If non-null, the bodies. </param>
/// <param name="p_decomposedForces">	[in,out] If non-null, the decomposed forces. </param>
/// <param name="p_bodyCount">		 	Number of bodies. </param>
///-------------------------------------------------------------------------------------------------
__global__ void MergeResults
		(Body* p_bodies, Vector2* p_decomposedForces, uint_t* p_startingPositions ,int p_bodyCount, float p_gTerm, float p_deltaT)
{
	uint_t globalIndex = GetGlobalIndex();
	Vector2 force(0), acceleration(0);

	if (globalIndex > 0)
	{
		//find the starting index from the forces array 
		uint_t startingIndex = threadIdx.x;
		uint_t jumpLength = p_bodyCount - 2;
		uint_t arrayIndex = threadIdx.x;
		uint_t remaningBodies = p_bodyCount - 1;

		//execute jump indices to get force aggregation
		for (int jumpIndex = startingIndex; jumpIndex > 0; jumpIndex--)
		{
			force += p_decomposedForces[arrayIndex];
			arrayIndex += jumpLength;
			jumpLength--, remaningBodies--;
		}

		arrayIndex = p_startingPositions[globalIndex];
		//execute linear summation of remaning force components 
		for (int index = 0; index < remaningBodies; index++, arrayIndex++)
		{
			force += p_decomposedForces[arrayIndex];
		}
	}
	//else if index  equal to 0 just traverse the first elements of the forces array
	else
	{
		for (int index = 0; index < p_bodyCount - 1; index++)
		{
			force += p_decomposedForces[index];
		}
	}

	//compute the new position
	acceleration = force * p_gTerm;
	Body *outcome = &p_bodies[globalIndex];

	p_bodies[globalIndex].Velocity += acceleration * p_deltaT;
	p_bodies[globalIndex].Position += outcome->Velocity * p_deltaT;
}

///-------------------------------------------------------------------------------------------------
/// <summary>
/// 	Trivial compute iteration.
/// 	Parallelise by spawning 1 thread per body 
/// </summary>
///
/// <remarks>	Adrian, 10/07/2017. </remarks>
///
/// <param name="p_bodies">		  	[in,out] If non-null, the bodies. </param>
/// <param name="p_updatedBodies">	[in,out] If non-null, the updated bodies. </param>
/// <param name="p_deltaT">		  	The delta t. </param>
/// <param name="p_bodyCount">	  	Number of bodies. </param>
/// <param name="p_gTerm">		  	The term. </param>
///-------------------------------------------------------------------------------------------------

__global__ void TrivialComputeIteration(Body* p_bodies, Body* p_updatedBodies, float p_deltaT, int p_bodyCount, float p_gTerm )
{
	int blockDimension = blockDim.x * blockDim.y * blockDim.z;
	int blockInnerIndex = threadIdx.x + blockDim.x * threadIdx.y;
	int globalIndex = blockInnerIndex + gridDim.x * blockIdx.y * blockDimension + blockDimension * blockIdx.x;

	if (globalIndex < p_bodyCount )
	{
		Vector2 force(0), acceleration(0);
		Body chosenBody = p_bodies[globalIndex];

		for (int index = 0; index < p_bodyCount; index++)
		{
			if (index == globalIndex) continue;
			CalculateForce(&chosenBody, &p_bodies[index], &force);
		}

		acceleration = force * p_gTerm;
		Body *outcome = &p_updatedBodies[globalIndex];

		p_updatedBodies[globalIndex].Velocity += acceleration * p_deltaT;
		p_updatedBodies[globalIndex].Position += outcome->Velocity * p_deltaT;
	}
}

///-------------------------------------------------------------------------------------------------
/// <summary>	
/// 			Creates parallelisation structure.
/// 			Creates the parallelisation structure through a greedy approach
/// </summary>
///
/// <remarks>	Adrian, 09/07/2017. </remarks>
///
/// <param name="p_bodyCount">	Number of bodies. </param>
///
/// <returns>	Returns dimensions of grid and block size. </returns>
///-------------------------------------------------------------------------------------------------
__host__ dim3 CreateParallelisationStructure(const int p_bodyCount)
{
	hipDeviceProp_t prop;
	//get device 0
	dim3 configuration;
	
	hipGetDeviceProperties(&prop, 0);
	
	//number of threads per block
	int maxNumberThreads = prop.maxThreadsPerBlock;
	//max number of blocks in the x direction
	int maxNumberBlocks = prop.maxGridSize[1];

	if (p_bodyCount < maxNumberThreads)
	{
		configuration.x = p_bodyCount;
		configuration.y = 1;
		configuration.z = 1;
	}
	else
	{
		int amountBlocks = std::ceilf(p_bodyCount / maxNumberThreads);
		
		configuration.x = maxNumberThreads;
		configuration.y = amountBlocks;
		configuration.z = 1;
	}

	return configuration;
}

///-------------------------------------------------------------------------------------------------
/// <summary>	
/// 	Main entry-point for this application. 
/// 	Switch between the two methods via the definition NORMAL_PARALLELISATION
/// </summary>
///
/// <remarks>	Adrian, 10/07/2017. </remarks>
///
/// <param name="argc">	Number of command-line arguments. </param>
/// <param name="argv">	Array of command-line argument strings. </param>
///
/// <returns>	Exit-code for the process - 0 for success, else an error code. </returns>
///-------------------------------------------------------------------------------------------------
int main(int argc, char* argv)
{
	
	hipSetDevice(0);
	hipDeviceSynchronize();
	hipDeviceSynchronize();

	int bodyCount = 0;
	bool success = false;

	float gTerm = 20.f;
	float deltaT = 0.1;

	Body *d_allBodies = nullptr;
	Body *d_allUpdatedBodies = nullptr;
	Vector2 *d_forcesMatrix = nullptr;
	uint_t* d_matrixIndex = nullptr;

	Body* allBodies = Core::System::IO::ParseBodiesInputFile(".//InputFiles//input_4096.txt", &bodyCount, &success);
	Body* allBodiesDump = new Body[bodyCount];

	if (!success)
	{
		std::cout << "Bodies not loaded properly!" << std::endl;
		Core::System::IO::Pause();
		return -1;
	}

	CheckForErrors(hipMalloc(&d_allBodies, sizeof(Body) * bodyCount));
	CheckForErrors(hipMalloc(&d_allUpdatedBodies, sizeof(Body) * bodyCount));
	CheckForErrors(hipMemcpy(d_allBodies, allBodies, bodyCount * sizeof(Body), hipMemcpyHostToDevice));
	CheckForErrors(hipMemcpy(d_allUpdatedBodies, allBodies, bodyCount * sizeof(Body), hipMemcpyHostToDevice));

	dim3 parallel = CreateParallelisationStructure(bodyCount);

	#ifdef NORMAL_PARALLELISATION
		for (int index = 0; index < 1000; index++)
		{
			Core::System::Util::WriteProgressMessage(index + 1);
			TrivialComputeIteration <<<parallel.y, parallel.x >>> (d_allBodies, d_allUpdatedBodies, deltaT, bodyCount, gTerm);
			hipDeviceSynchronize();
			CheckForErrors(hipMemcpy(allBodiesDump, d_allUpdatedBodies, bodyCount * sizeof(Body), hipMemcpyDeviceToHost));
			CheckForErrors(hipMemcpy(d_allBodies, d_allUpdatedBodies, bodyCount * sizeof(Body), hipMemcpyDeviceToDevice));
			//dump data to file
			Core::System::IO::WriteBodiesToFile(index, allBodiesDump, bodyCount);
		}
	#else
		
		int matrixSize = (bodyCount * bodyCount) - bodyCount / 2;

		CheckForErrors(hipMalloc(&d_forcesMatrix, matrixSize * sizeof(float)));
		CheckForErrors(hipMalloc(&d_matrixIndex, matrixSize * sizeof(int)));

		for (int index = 0; index < 1000; index++)
		{
			Core::System::IO::WriteProgressMessage(index + 1);
			ComputeIterationOptimised <<<parallel.y, parallel.x>>>(d_allBodies, d_forcesMatrix, d_matrixIndex, bodyCount, gTerm, deltaT);
			hipDeviceSynchronize();
			MergeResults <<<parallel.y, parallel.x>>>(d_allBodies, d_forcesMatrix, d_matrixIndex, bodyCount, 20.f, 0.5f);
			CheckForErrors(hipMemcpy(allBodiesDump, d_allBodies, bodyCount * sizeof(Body), hipMemcpyDeviceToHost));
			Core::System::IO::WriteBodiesToFile(index, allBodiesDump, bodyCount);
		}

	#endif 
	hipFree(d_allBodies);
	hipFree(d_allUpdatedBodies);

	delete[] allBodies;
	delete[] allBodiesDump;

	Core::System::IO::Pause();

   return 0;
}


